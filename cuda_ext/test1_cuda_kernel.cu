#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
/*
    define your own cuda extension.

    This example is just add N to the original image.
*/
namespace {
template <typename scalar_t>
__global__ void test1_cuda_kernel(
    scalar_t* __restrict__ image,
    size_t N,
    size_t batch_size,
    size_t channel,
    size_t image_height,
    size_t image_width) {

        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x * gridDim.x;

        while(idx < batch_size*channel*image_height*image_width) {
            image[idx] = image[idx] + N;
            idx += num_threads;
        }
    }
}

at::Tensor test1_cuda(
        at::Tensor image,
        size_t N) {
    const auto batch_size = image.size(0);
    const auto channel = image.size(1);
    const auto image_height = image.size(2);
    const auto image_width = image.size(3);

    const int threads = 32;
    const dim3 blocks ((batch_size * channel - 1) / threads + 1);

     // 注意, AT_DISPATCH_FLOATING_TYPES的第2个参数必须和所在函数体的名称一样! 否则会就无法dispatch.
     AT_DISPATCH_FLOATING_TYPES(image.type(), "test1_cuda", ([&] {
      test1_cuda_kernel<scalar_t><<<blocks, threads>>>(
          image.data<scalar_t>(),
          N,
          batch_size,
          channel,
          image_height,
          image_width);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
            printf("Error in test1: %s\n", hipGetErrorString(err));
    return image;
}